#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "vector.h"
#include "config.h"
#include "planets.h"
#include "compute.h"

// represents the objects in the system.  Global variables
vector3 *hVel, *d_hVel;
vector3 *hPos, *d_hPos;
vector3 **accels, **d_accels;
vector3 **h_accels; // remove this line A0
double *mass, *d_mass;

//initHostMemory: Create storage for numObjects entities in our system
//Parameters: numObjects: number of objects to allocate
//Returns: None
//Side Effects: Allocates memory in the hVel, hPos, and mass global variables
void initHostMemory(int numObjects)
{
	hVel = (vector3 *)malloc(sizeof(vector3) * numObjects);
	hPos = (vector3 *)malloc(sizeof(vector3) * numObjects);
	mass = (double *)malloc(sizeof(double) * numObjects);
}

//freeHostMemory: Free storage allocated by a previous call to initHostMemory
//Parameters: None
//Returns: None
//Side Effects: Frees the memory allocated to global variables hVel, hPos, and mass.
void freeHostMemory()
{
	free(hVel);
	free(hPos);
	free(mass);
}

void initDeviceMemory() {
	hipError_t err;

	accels = (vector3**)malloc(sizeof(vector3*) * NUMENTITIES);
	h_accels = (vector3**)malloc(sizeof(vector3*) * NUMENTITIES); // remove this line A0
	for(int i = 0; i < NUMENTITIES; i++) {
		err = hipMalloc(&accels[i], sizeof(vector3) * NUMENTITIES);
		if(hipSuccess != err) {
			printf("Error hipMalloc accels[%d]: %s\n", i, hipGetErrorString(err));
		}
		h_accels[i] = (vector3*)malloc(sizeof(vector3) * NUMENTITIES); // remove this line A0
	}
	err = hipMalloc(&d_accels, sizeof(vector3*) * NUMENTITIES);
	if(hipSuccess != err) {
		printf("Error hipMalloc d_accels: %s\n", hipGetErrorString(err));
	}
	err = hipMemcpy(d_accels, accels, sizeof(vector3*) * NUMENTITIES, hipMemcpyHostToDevice);
	if(hipSuccess != err) {
		printf("Error hipMemcpy: %s\n", hipGetErrorString(err));
	}

	err = hipMalloc(&d_hVel, sizeof(vector3) * NUMENTITIES);
	if(hipSuccess != err) {
		printf("Error hipMalloc d_hVel: %s\n", hipGetErrorString(err));
	}
	err = hipMemcpy(d_hVel, hVel, sizeof(vector3) * NUMENTITIES, hipMemcpyHostToDevice);
	if(hipSuccess != err) {
		printf("Error hipMemcpy d_hVel: %s\n", hipGetErrorString(err));
	}

	err = hipMalloc(&d_hPos, sizeof(vector3) * NUMENTITIES);
	if(hipSuccess != err) {
		printf("Error hipMalloc d_hPos: %s\n", hipGetErrorString(err));
	}
	err = hipMemcpy(d_hPos, hPos, sizeof(vector3) * NUMENTITIES, hipMemcpyHostToDevice);
	if(hipSuccess != err) {
		printf("Error hipMemcpy d_hPos: %s\n", hipGetErrorString(err));
	}

	err = hipMalloc(&d_mass, sizeof(double) * NUMENTITIES);
	if(hipSuccess != err) {
		printf("Error hipMalloc d_mass: %s\n", hipGetErrorString(err));
	}
	err = hipMemcpy(d_mass, mass, sizeof(double) * NUMENTITIES, hipMemcpyHostToDevice);
	if(hipSuccess != err) {
		printf("Error hipMemcpy d_mass: %s\n", hipGetErrorString(err));
	}
}

void freeDeviceMemory() {
	for(int i = 0; i < NUMENTITIES; i++) {
		hipFree(accels[i]);
	}
	hipFree(d_accels);
	free(accels);

	hipFree(d_hVel);
	hipFree(d_hPos);
	hipFree(d_mass);
	free(h_accels); // remove this line A0
}

void getDeviceMemory() {
	hipMemcpy(hVel, d_hVel, sizeof(vector3) * NUMENTITIES, hipMemcpyDeviceToHost);
	hipMemcpy(hPos, d_hPos, sizeof(vector3) * NUMENTITIES, hipMemcpyDeviceToHost);
}

//planetFill: Fill the first NUMPLANETS+1 entries of the entity arrays with an estimation
//				of our solar system (Sun+NUMPLANETS)
//Parameters: None
//Returns: None
//Fills the first 8 entries of our system with an estimation of the sun plus our 8 planets.
void planetFill(){
	int i,j;
	double data[][7]={SUN,MERCURY,VENUS,EARTH,MARS,JUPITER,SATURN,URANUS,NEPTUNE};
	for (i=0;i<=NUMPLANETS;i++){
		for (j=0;j<3;j++){
			hPos[i][j]=data[i][j];
			hVel[i][j]=data[i][j+3];
		}
		mass[i]=data[i][6];
	}
}

//randomFill: FIll the rest of the objects in the system randomly starting at some entry in the list
//Parameters: 	start: The index of the first open entry in our system (after planetFill).
//				count: The number of random objects to put into our system
//Returns: None
//Side Effects: Fills count entries in our system starting at index start (0 based)
void randomFill(int start, int count)
{
	int i, j = start;
	for (i = start; i < start + count; i++)
	{
		for (j = 0; j < 3; j++)
		{
			hVel[i][j] = (double)rand() / RAND_MAX * MAX_DISTANCE * 2 - MAX_DISTANCE;
			hPos[i][j] = (double)rand() / RAND_MAX * MAX_VELOCITY * 2 - MAX_VELOCITY;
			mass[i] = (double)rand() / RAND_MAX * MAX_MASS;
		}
	}
}

//printSystem: Prints out the entire system to the supplied file
//Parameters: 	handle: A handle to an open file with write access to prnt the data to
//Returns: 		none
//Side Effects: Modifies the file handle by writing to it.
void printSystem(FILE* handle){
	int i,j;
	for (i=0;i<NUMENTITIES;i++){
		fprintf(handle,"pos=(");
		for (j=0;j<3;j++){
			fprintf(handle,"%lf,",hPos[i][j]);
		}
		printf("),v=(");
		for (j=0;j<3;j++){
			fprintf(handle,"%lf,",hVel[i][j]);
		}
		fprintf(handle,"),m=%lf\n",mass[i]);
	}
}

int main(int argc, char **argv)
{
	clock_t t0=clock();
	int t_now;
	//srand(time(NULL));
	srand(1234);
	initHostMemory(NUMENTITIES);
	planetFill();
	randomFill(NUMPLANETS + 1, NUMASTEROIDS);
	//now we have a system.
	#ifdef DEBUG
	printSystem(stdout);
	#endif

	initDeviceMemory();

	// for (t_now=0;t_now<DURATION;t_now+=INTERVAL){
	// 	compute();
	// }
	compute();

	getDeviceMemory();

	clock_t t1=clock()-t0;
#ifdef DEBUG
	printSystem(stdout);
#endif
	printf("This took a total time of %f seconds\n",(double)t1/CLOCKS_PER_SEC);

	freeDeviceMemory();
	freeHostMemory();
}
